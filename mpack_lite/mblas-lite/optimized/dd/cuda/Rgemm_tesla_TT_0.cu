#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2011
 *      RIKEN
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */
/*
  Contributed by Takao, Yasuyoshi and Nakata, Maho, 2010-2011
*/

//for alpha*A'*B' + beta*C
__global__ void Rgemm_tesla_TT_0(dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta)
{
    int i, j;
    dd_real c_val0;
    dd_real c_val1;
    int iAb, jAb, A_i, A_j, Ab_i, Ab_j;
    int iBb, jBb, B_i, B_j, Bb_i, Bb_j;
    int iCb, jCb, C_i, C_j;
    dd_real Aval;
    int Lp;
    dd_real regA;
    dd_real regB0;
    dd_real regB1;
    dd_real temp;

    __shared__ dd_real Ab[Bm][Bk + 1];
    __shared__ dd_real Bb[2][Bn][Bk + 1];

    Ab_j = threadIdx.x; //exchange x for y for coalescing
    Bb_i = threadIdx.y; //exchange x for y for coalescing

    iAb = blockIdx.x;
    A_i = blockDim.x * iAb + threadIdx.y; //exchange x for y for coalescing

    //load first data of A from global memory into register
    A_j = blockDim.y * 0 + threadIdx.x; //exchange x for y for coalescing
    regA = fetch_x_A(A_i * lda + A_j);

    //load first data of B from global memory into register
    iBb = 0;
    B_i = blockDim.x * iBb + threadIdx.y; //exchange x for y for coalescing

    jBb = blockIdx.y * Gn + 0;
    B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
    regB0 = fetch_x_B(B_i * ldb + B_j);

    jBb = blockIdx.y * Gn + 1;
    B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
    regB1 = fetch_x_B(B_i * ldb + B_j);

    c_val0.x[0] = c_val0.x[1] = c_val1.x[0] = c_val1.x[1] = 0.0;

    for (i = 0; i < k / Bk; i++) {
        Ab_i = threadIdx.y; //exchange x for y for coalescing
	// load data into Ab (in shared memory) from register
	Ab[Ab_i][Ab_j] = regA;

	// load data into Bb (in shared memory) from register
        Bb_j = threadIdx.x + 0 * blockDim.y; //exchange x for y for coalescing
	Bb[0][Bb_j][Bb_i] = regB0;
	Bb[1][Bb_j][Bb_i] = regB1;

	// syncronize in the block
	__syncthreads();

	// update C value
	Lp = Bk;

        Ab_i = threadIdx.x; //recover Ab_i for coalescing
        Bb_j = threadIdx.y + 0 * blockDim.y; //recover Bb_j for coalescing

#pragma unroll
	for (j = 0; j < Lp; j++) {
	    //take advantage of speed difference between register and smem
	    Aval = Ab[Ab_i][j];
	    dd_mad(c_val0, Aval, Bb[0][Bb_j][j]);
	    dd_mad(c_val1, Aval, Bb[1][Bb_j][j]);
	}

	// load next data of A from global memory into register
	jAb = i + 1;
        A_j = blockDim.y * jAb + threadIdx.x; //exchange x for y for coalescing
	regA = fetch_x_A(A_i * lda + A_j);

	// load next data of B from global memory into register
	iBb = i + 1;
        B_i = blockDim.x * iBb + threadIdx.y; //exchange x for y for coalescing

	jBb = blockIdx.y * Gn + 0;
        B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
	regB0 = fetch_x_B(B_i * ldb + B_j);

	jBb = blockIdx.y * Gn + 1;
        B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
	regB1 = fetch_x_B(B_i * ldb + B_j);

	__syncthreads();
    }

    dd_mul(c_val0, alpha, c_val0);
    dd_mul(c_val1, alpha, c_val1);

    // update Cdev (global memory)
    iCb = blockIdx.x;
    C_i = blockDim.x * iCb + threadIdx.x;

    jCb = blockIdx.y * Gn + 0;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp = Cdev[C_i + C_j * ldc];
    dd_mul(beta, temp, temp);
    dd_add(temp, c_val0, Cdev[C_i + C_j * ldc]);

    jCb = blockIdx.y * Gn + 1;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp = Cdev[C_i + C_j * ldc];
    dd_mul(beta, temp, temp);
    dd_add(temp, c_val1, Cdev[C_i + C_j * ldc]);

}
