#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2011
 *      RIKEN
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */
/*
  Contributed by Takao, Yasuyoshi and Nakata, Maho, 2010-2011
*/

//for alpha*A'*B' + beta*C
__global__ void Rgemm_tesla_TT_p(dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta)
{
    int i, j;
    dd_real c_val0;
    dd_real c_val1;
    int iAb, jAb, A_i, A_j, Ab_i, Ab_j;
    int iBb, jBb, B_i, B_j, Bb_i, Bb_j;
    int iCb, jCb, C_i, C_j;
    dd_real Aval;
    int Lp;
    dd_real regA;
    dd_real regB0;
    dd_real regB1;
    dd_real temp0;
    dd_real temp1;

    __shared__ dd_real Ab[Bm][Bk + 1];
    __shared__ dd_real Bb[2][Bn][Bk + 1];

    Ab_j = threadIdx.x; //exchange x for y for coalescing
    Bb_i = threadIdx.y; //exchange x for y for coalescing

    iAb = blockIdx.x;
    A_i = blockDim.x * iAb + threadIdx.y; //exchange x for y for coalescing

    //load first data of A from global memory into register
    A_j = blockDim.y * 0 + threadIdx.x; //exchange x for y for coalescing
    regA = fetch_x_A(min(A_i, (int) (m - 1)) * lda + min(A_j, (int) (k - 1)));

    //load first data of B from global memory into register
    iBb = 0;
    B_i = blockDim.x * iBb + threadIdx.y; //exchange x for y for coalescing

    jBb = blockIdx.y * Gn + 0;
    B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
    regB0 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

    jBb = blockIdx.y * Gn + 1;
    B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
    regB1 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

    // get initial Cdev data
    iCb = blockIdx.x;
    C_i = blockDim.x * iCb + threadIdx.x;

    jCb = blockIdx.y * Gn + 0;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp0 = Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc];

    jCb = blockIdx.y * Gn + 1;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp1 = Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc];

    c_val0.x[0] = c_val0.x[1] = c_val1.x[0] = c_val1.x[1] = 0.0;

    for (i = 0; i < k / Bk + (k % Bk == 0 ? 0 : 1); i++) {
        Ab_i = threadIdx.y; //exchange x for y for coalescing
	// load data into Ab (in shared memory) from register
	Ab[Ab_i][Ab_j] = regA;

	// load data into Bb (in shared memory) from register
        Bb_j = threadIdx.x + 0 * blockDim.y; //exchange x for y for coalescing
	Bb[0][Bb_j][Bb_i] = regB0;
	Bb[1][Bb_j][Bb_i] = regB1;

	// syncronize in the block
	__syncthreads();

	// update C value
	Lp = (k % Bk == 0 ? Bk : (i < k / Bk ? Bk : k % Bk));

        Ab_i = threadIdx.x; //recover Ab_i for coalescing
        Bb_j = threadIdx.y + 0 * blockDim.y; //recover Bb_j for coalescing

#pragma unroll
	for (j = 0; j < Lp; j++) {
	    //take advantage of speed difference between register and smem
	    Aval = Ab[Ab_i][j];
	    dd_mad(c_val0, Aval, Bb[0][Bb_j][j]);
	    dd_mad(c_val1, Aval, Bb[1][Bb_j][j]);
	}

	// load next data of A from global memory into register
	jAb = i + 1;
        A_j = blockDim.y * jAb + threadIdx.x; //exchange x for y for coalescing
	regA = fetch_x_A(min(A_i, (int) (m - 1)) * lda + min(A_j, (int) (k - 1)));

	// load next data of B from global memory into register
	iBb = i + 1;
        B_i = blockDim.x * iBb + threadIdx.y; //exchange x for y for coalescing

	jBb = blockIdx.y * Gn + 0;
        B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
	regB0 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

	jBb = blockIdx.y * Gn + 1;
        B_j = blockDim.y * jBb + threadIdx.x; //exchange x for y for coalescing
	regB1 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

	__syncthreads();
    }

    dd_mul(c_val0, alpha, c_val0);
    dd_mul(c_val1, alpha, c_val1);

    jCb = blockIdx.y * Gn + 0;
    C_j = blockDim.y * jCb + threadIdx.y;
    dd_mul(beta, temp0, temp0);
    dd_add(temp0, c_val0, Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc]);

    jCb = blockIdx.y * Gn + 1;
    C_j = blockDim.y * jCb + threadIdx.y;
    dd_mul(beta, temp1, temp1);
    dd_add(temp1, c_val1, Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc]);

}
