#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2011
 *      RIKEN
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */
/*
  Contributed by Takao, Yasuyoshi and Nakata, Maho, 2010-2011
*/

//for alpha*A'*B' + beta*C
__global__ void Rgemm_fermi_TT_p(dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta)
{
    int i, j;
    dd_real c_val0;
    dd_real c_val1;
    dd_real c_val2;
    dd_real c_val3;
    int iAb, jAb, A_i, A_j, Ab_i, Ab_j;
    int iBb, jBb, B_i, B_j, Bb_i, Bb_j;
    int iCb, jCb, C_i, C_j;
    dd_real Aval;
    int Lp;
    dd_real regA;
    dd_real regB0;
    dd_real regB1;
    dd_real regB2;
    dd_real regB3;
    dd_real temp0;
    dd_real temp1;
    dd_real temp2;
    dd_real temp3;

    __shared__ dd_real Ab[Bm][Bk + 1];
    __shared__ dd_real Bb[4][Bn][Bk + 1];

    Ab_i = threadIdx.x;
    Ab_j = threadIdx.y;
    Bb_i = threadIdx.x;

    iAb = blockIdx.x;
    A_i = blockDim.x * iAb + threadIdx.x;

    Bb_j = threadIdx.y + 0 * blockDim.y;

    //load first data of A from global memory into register
    A_j = blockDim.y * 0 + threadIdx.y;
    regA = fetch_x_A(min(A_i, (int) (m - 1)) * lda + min(A_j, (int) (k - 1)));

    //load first data of B from global memory into register
    iBb = 0;
    B_i = blockDim.x * iBb + threadIdx.x;

    jBb = blockIdx.y * Gn + 0;
    B_j = blockDim.y * jBb + threadIdx.y;
    regB0 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

    jBb = blockIdx.y * Gn + 1;
    B_j = blockDim.y * jBb + threadIdx.y;
    regB1 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

    jBb = blockIdx.y * Gn + 2;
    B_j = blockDim.y * jBb + threadIdx.y;
    regB2 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

    jBb = blockIdx.y * Gn + 3;
    B_j = blockDim.y * jBb + threadIdx.y;
    regB3 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

    // get initial Cdev data
    iCb = blockIdx.x;
    C_i = blockDim.x * iCb + threadIdx.x;

    jCb = blockIdx.y * Gn + 0;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp0 = Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc];

    jCb = blockIdx.y * Gn + 1;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp1 = Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc];

    jCb = blockIdx.y * Gn + 2;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp2 = Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc];

    jCb = blockIdx.y * Gn + 3;
    C_j = blockDim.y * jCb + threadIdx.y;
    temp3 = Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc];

    c_val0.x[0] = c_val0.x[1] = c_val1.x[0] = c_val1.x[1] = 0.0;
    c_val2.x[0] = c_val2.x[1] = c_val3.x[0] = c_val3.x[1] = 0.0;

    for (i = 0; i < k / Bk + (k % Bk == 0 ? 0 : 1); i++) {
	// load data into Ab (in shared memory) from register
	Ab[Ab_i][Ab_j] = regA;

	// load data into Bb (in shared memory) from register
	Bb[0][Bb_j][Bb_i] = regB0;
	Bb[1][Bb_j][Bb_i] = regB1;
	Bb[2][Bb_j][Bb_i] = regB2;
	Bb[3][Bb_j][Bb_i] = regB3;

	// syncronize in the block
	__syncthreads();

	// update C value
	Lp = (k % Bk == 0 ? Bk : (i < k / Bk ? Bk : k % Bk));

#pragma unroll
	for (j = 0; j < Lp; j++) {
	    //take advantage of speed difference between register and smem
	    Aval = Ab[Ab_i][j];
	    dd_mad(c_val0, Aval, Bb[0][Bb_j][j]);
	    dd_mad(c_val1, Aval, Bb[1][Bb_j][j]);
	    dd_mad(c_val2, Aval, Bb[2][Bb_j][j]);
	    dd_mad(c_val3, Aval, Bb[3][Bb_j][j]);
	}

	// load next data of A from global memory into register
	jAb = i + 1;
	A_j = blockDim.y * jAb + threadIdx.y;
	regA = fetch_x_A(min(A_i, (int) (m - 1)) * lda + min(A_j, (int) (k - 1)));

	// load next data of B from global memory into register
	iBb = i + 1;
	B_i = blockDim.x * iBb + threadIdx.x;

	jBb = blockIdx.y * Gn + 0;
	B_j = blockDim.y * jBb + threadIdx.y;
	regB0 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

	jBb = blockIdx.y * Gn + 1;
	B_j = blockDim.y * jBb + threadIdx.y;
	regB1 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

	jBb = blockIdx.y * Gn + 2;
	B_j = blockDim.y * jBb + threadIdx.y;
	regB2 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

	jBb = blockIdx.y * Gn + 3;
	B_j = blockDim.y * jBb + threadIdx.y;
	regB3 = fetch_x_B(min(B_i, (int) (k - 1)) * ldb + min(B_j, (int) (n - 1)));

	__syncthreads();
    }

    dd_mul(c_val0, alpha, c_val0);
    dd_mul(c_val1, alpha, c_val1);
    dd_mul(c_val2, alpha, c_val2);
    dd_mul(c_val3, alpha, c_val3);

    jCb = blockIdx.y * Gn + 0;
    C_j = blockDim.y * jCb + threadIdx.y;
    dd_mul(beta, temp0, temp0);
    dd_add(temp0, c_val0, Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc]);

    jCb = blockIdx.y * Gn + 1;
    C_j = blockDim.y * jCb + threadIdx.y;
    dd_mul(beta, temp1, temp1);
    dd_add(temp1, c_val1, Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc]);

    jCb = blockIdx.y * Gn + 2;
    C_j = blockDim.y * jCb + threadIdx.y;
    dd_mul(beta, temp2, temp2);
    dd_add(temp2, c_val2, Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc]);

    jCb = blockIdx.y * Gn + 3;
    C_j = blockDim.y * jCb + threadIdx.y;
    dd_mul(beta, temp3, temp3);
    dd_add(temp3, c_val3, Cdev[min(C_i, (int) (m - 1)) + min(C_j, (int) (n - 1)) * ldc]);
}
