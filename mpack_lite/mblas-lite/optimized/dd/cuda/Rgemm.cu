/*
 * Copyright (c) 2010-2011
 *      RIKEN
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */
/*
  Contributed by Takao, Yasuyoshi and Nakata, Maho, 2010-2011
*/
/*
Based on http://www.netlib.org/blas/dgemm.f
Rgemm performs one of the matrix-matrix operations
 C := alpha*op(A)*op(B) + beta*C,
where op(X) is one of
 op(X) = X or op(X) = X',
alpha and beta are scalars, and A, B and C are matrices, with op( A )
an m by k matrix, op(B) a k by n matrix and C an m by n matrix.
*/

#include <iostream>
#include <stdio.h>
#include "dd_real_cuda.h"
#include <mpack_config.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void Rgemm_fermi(const char *transa, const char *transb, mpackint m, mpackint n, mpackint k, dd_real alpha, dd_real * A, mpackint lda, dd_real * B, mpackint ldb, dd_real beta, dd_real * C, mpackint ldc);

void Rgemm_tesla(const char *transa, const char *transb, mpackint m, mpackint n, mpackint k, dd_real alpha, dd_real * A, mpackint lda, dd_real * B, mpackint ldb, dd_real beta, dd_real * C, mpackint ldc);

void Rgemm(const char *transa, const char *transb, mpackint m, mpackint n, mpackint k, dd_real alpha, dd_real * A, mpackint lda, dd_real * B, mpackint ldb, dd_real beta, dd_real * C, mpackint ldc)
{
//just call fermi version
    Rgemm_fermi(transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

//if tesla version
//  Rgemm_tesla(transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    return;
}
