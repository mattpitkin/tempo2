#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2011
 *      RIKEN
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */
/*
  Contributed by Takao, Yasuyoshi and Nakata, Maho, 2010-2011
*/
/*
Based on http://www.netlib.org/blas/dgemm.f
Rgemm performs one of the matrix-matrix operations
 C := alpha*op(A)*op(B) + beta*C,
where op(X) is one of
 op(X) = X or op(X) = X',
alpha and beta are scalars, and A, B and C are matrices, with op( A )
an m by k matrix, op(B) a k by n matrix and C an m by n matrix.
*/

#include <iostream>
#include <stdio.h>
#include "dd_real_cuda.h"
#include <mpack_config.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

mpackint Mlsame_dd(const char *a, const char *b);
void Mxerbla_dd(const char *srname, int info);

// define texture memory
texture < int4, 1 > tex_x_double_A;
texture < int4, 1 > tex_x_double_B;

// matrix block size
#define Bm  (16)
#define Bk  (16)
#define Bn  (16)
#define Gn   (2)

static
__inline__ __device__ dd_real fetch_x_A(const int &i)
{
    register int4 v = tex1Dfetch(tex_x_double_A, i);
    register
    dd_real r;
    r.x[0] = __hiloint2double(v.y, v.x);
    r.x[1] = __hiloint2double(v.w, v.z);
    return r;
}

static
__inline__ __device__ dd_real fetch_x_B(const int &i)
{
    register int4 v = tex1Dfetch(tex_x_double_B, i);
    register
    dd_real r;
    r.x[0] = __hiloint2double(v.y, v.x);
    r.x[1] = __hiloint2double(v.w, v.z);
    return r;
}

//for alpha*A*B + beta
__global__ void Rgemm_tesla_NN_0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void Rgemm_tesla_NN_p (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

//for alpha*A'*B + beta
__global__ void Rgemm_tesla_TN_0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void Rgemm_tesla_TN_p (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

//for alpha*A*B' + beta
__global__ void Rgemm_tesla_NT_0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void Rgemm_tesla_NT_p (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

//for alpha*A'*B' + beta
__global__ void Rgemm_tesla_TT_0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void Rgemm_tesla_TT_p (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

void Is_cuda_Rgemm_error(hipError_t rc, const char *mes, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc);

#include <Rgemm_tesla_NN_0.cu>
#include <Rgemm_tesla_NN_p.cu>
#include <Rgemm_tesla_TN_0.cu>
#include <Rgemm_tesla_TN_p.cu>
#include <Rgemm_tesla_NT_0.cu>
#include <Rgemm_tesla_NT_p.cu>
#include <Rgemm_tesla_TT_0.cu>
#include <Rgemm_tesla_TT_p.cu>

void Rgemm_tesla_cuda(const char *transa, const char *transb, mpackint m, mpackint n, mpackint k, dd_real alpha, dd_real * Adev, mpackint lda, dd_real * Bdev, mpackint ldb, dd_real beta, dd_real * Cdev, mpackint ldc)
{
    mpackint nota, notb;
    hipError_t rc;

    nota = Mlsame_dd(transa, "N");
    notb = Mlsame_dd(transb, "N");

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindSigned);
    // bind texture memory
    rc = hipBindTexture(0, tex_x_double_A, Adev, channelDesc);
        Is_cuda_Rgemm_error(rc, "could not bind to texture A", m, n, k, lda, ldb, ldc);

    rc = hipBindTexture(0, tex_x_double_B, Bdev, channelDesc);
        Is_cuda_Rgemm_error(rc, "could not bind to texture B", m, n, k, lda, ldb, ldc);

    if (notb) {
        if (nota) {
	    //Form C := alpha*A*B + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                Rgemm_tesla_NN_0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                Rgemm_tesla_NN_p <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        } else {
	    //Form C := alpha*A'*B + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                Rgemm_tesla_TN_0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                Rgemm_tesla_TN_p <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        }
    } else {
        if (nota) {
	    //Form  C := alpha*A*B' + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                Rgemm_tesla_NT_0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                Rgemm_tesla_NT_p <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        } else {
	    //Form  C := alpha*A'*B' + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                Rgemm_tesla_TT_0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                Rgemm_tesla_TT_p <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        }
    }
    //unbind texture
    rc = hipUnbindTexture(tex_x_double_A);
        Is_cuda_Rgemm_error(rc, "hipUnbindTexture A error", m, n, k, lda, ldb, ldc);
    rc = hipUnbindTexture(tex_x_double_B);
        Is_cuda_Rgemm_error(rc, "hipUnbindTexture B error", m, n, k, lda, ldb, ldc);
    hipDeviceSynchronize();
}

void Rgemm_tesla(const char *transa, const char *transb, mpackint m, mpackint n, mpackint k, dd_real alpha, dd_real * A, mpackint lda, dd_real * B, mpackint ldb, dd_real beta, dd_real * C, mpackint ldc)
{
    mpackint i, j, nota, notb, nrowa, nrowb, ncola, info;
    dd_real temp, Zero, One;
    hipError_t rc;

    dd_set(Zero, 0.0, 0.0);
    dd_set(One, 1.0, 0.0);

    nota = Mlsame_dd(transa, "N");
    notb = Mlsame_dd(transb, "N");
    if (nota) {
	nrowa = m;
	ncola = k;
    } else {
	nrowa = k;
	ncola = m;
    }
    if (notb) {
	nrowb = k;
    } else {
	nrowb = n;
    }
//Test the input parameters.
    info = 0;
    if (!nota && (!Mlsame_dd(transa, "C"))
	&& (!Mlsame_dd(transa, "T")))
	info = 1;
    else if (!notb && (!Mlsame_dd(transb, "C"))
	     && (!Mlsame_dd(transb, "T")))
	info = 2;
    else if (m < 0)
	info = 3;
    else if (n < 0)
	info = 4;
    else if (k < 0)
	info = 5;
    else if (lda < std::max((mpackint) 1, nrowa))
	info = 8;
    else if (ldb < std::max((mpackint) 1, nrowb))
	info = 10;
    else if (ldc < std::max((mpackint) 1, m))
	info = 13;
    if (info != 0) {
	Mxerbla_dd("Rgemm ", info);
	return;
    }
    //Quick return if possible.
    if ((m == 0)
	|| (n == 0)
	|| ((dd_eq(alpha, Zero)
	     || (k == 0))
	    && dd_eq(beta, One))) {
	return;
    }

    //allocate device memory for GPU
    dd_real *Adev, *Bdev, *Cdev;
    int size_A, size_B, size_C;
    if (nota)
	size_A = lda * k - (lda - m);
    else
	size_A = lda * m - (lda - k);
    if (notb)
	size_B = ldb * n - (ldb - k);
    else
	size_B = ldb * k - (ldb - n);
    size_C = ldc * n - (ldc - m);
    rc = hipMalloc((void **) &Adev, size_A * sizeof(dd_real));
        Is_cuda_Rgemm_error(rc, "hipMalloc A error", m, n, k, lda, ldb, ldc);
    rc = hipMalloc((void **) &Bdev, size_B * sizeof(dd_real));
        Is_cuda_Rgemm_error(rc, "hipMalloc B error", m, n, k, lda, ldb, ldc);
    rc = hipMalloc((void **) &Cdev, size_C * sizeof(dd_real));
        Is_cuda_Rgemm_error(rc, "hipMalloc C error", m, n, k, lda, ldb, ldc);
    rc = hipMemcpy(Adev, A, size_A * sizeof(dd_real), hipMemcpyHostToDevice);
        Is_cuda_Rgemm_error(rc, "hipMemcpy A error", m, n, k, lda, ldb, ldc);
    rc = hipMemcpy(Bdev, B, size_B * sizeof(dd_real), hipMemcpyHostToDevice);
        Is_cuda_Rgemm_error(rc, "hipMemcpy B error", m, n, k, lda, ldb, ldc);
    rc = hipMemcpy(Cdev, C, size_C * sizeof(dd_real), hipMemcpyHostToDevice);
        Is_cuda_Rgemm_error(rc, "hipMemcpy C error", m, n, k, lda, ldb, ldc);

//And when alpha == 0.0
    if (dd_eq(alpha, Zero)) {
	if (dd_eq(beta, Zero)) {
	    for (j = 0; j < n; j++) {
		for (i = 0; i < m; i++) {
		    C[i + j * ldc] = Zero;
		}
	    }
	} else {
	    for (j = 0; j < n; j++) {
		for (i = 0; i < m; i++) {
		    dd_mul_host(beta, C[i + j * ldc], C[i + j * ldc]);
		}
	    }
	}
	return;
    }

    Rgemm_tesla_cuda(transa, transb, m, n, k, alpha, Adev, lda, Bdev, ldb, beta, Cdev, ldc);

    rc = hipMemcpy(C, Cdev, size_C * sizeof(dd_real), hipMemcpyDeviceToHost);
        Is_cuda_Rgemm_error(rc, "hipMemcpy C error", m, n, k, lda, ldb, ldc);
    rc = hipFree(Adev);
        Is_cuda_Rgemm_error(rc, "hipFree A error", m, n, k, lda, ldb, ldc);
    rc = hipFree(Bdev);
        Is_cuda_Rgemm_error(rc, "hipFree B error", m, n, k, lda, ldb, ldc);
    rc = hipFree(Cdev);
        Is_cuda_Rgemm_error(rc, "hipFree C error", m, n, k, lda, ldb, ldc);
    return;
}
